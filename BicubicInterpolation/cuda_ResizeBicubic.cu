#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "serial_ResizeBicubic.h"

using namespace std;

// for CUDA error checking
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << " : " << hipGetErrorString(err) << endl; \
            exit(err); \
        } \
    } while (0)

// Refer to bicubicKernal in bicubicKernel.cpp
__device__ float cuda_bicubicKernel(float d) {
    d = fabsf(d);
    if (d <= 1.0f) {
        return (1.5f * d * d * d - 2.5f * d * d + 1.0f);
    }
    else if (d <= 2.0f) {
        return (-0.5f * d * d * d + 2.5f * d * d - 4.0f * d + 2.0f);
    }
    return 0.0f;
}

// Refer to cuda_getPixelValue in bicubicKernel.cpp
__device__ float cuda_getPixelValue(unsigned char* image, int width, int height, int channels, int x, int y, int c) {
    x = max(0, min(x, width - 1));
    y = max(0, min(y, height - 1));
    return image[(y * width + x) * channels + c];
}

// CUDA kernel for bicubic resizing
__global__ void cuda_ResizeBicubicKernel(unsigned char* src, int srcWidth, int srcHeight, int channels,
    unsigned char* dst, int dstWidth, int dstHeight, float scaleX, float scaleY) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < dstWidth && y < dstHeight) {
        for (int c = 0; c < channels; ++c) {
            float srcX = x * scaleX;
            float srcY = y * scaleY;
            int x1 = (int)srcX;
            int y1 = (int)srcY;

            float result = 0.0f;
            for (int m = -1; m <= 2; ++m) {
                for (int n = -1; n <= 2; ++n) {
                    float weight = cuda_bicubicKernel(srcX - (x1 + n)) * cuda_bicubicKernel(srcY - (y1 + m));
                    result += cuda_getPixelValue(src, srcWidth, srcHeight, channels, x1 + n, y1 + m, c) * weight;
                }
            }

            dst[(y * dstWidth + x) * channels + c] = min(max((int)result, 0), 255);
        }
    }
}

// Function to resize the image on the GPU
void cuda_ResizeBicubic(unsigned char* src, int srcWidth, int srcHeight, int channels,
    unsigned char* dst, int dstWidth, int dstHeight) {
    unsigned char* d_src;
    unsigned char* d_dst;
    size_t srcSize = srcWidth * srcHeight * channels * sizeof(unsigned char);
    size_t dstSize = dstWidth * dstHeight * channels * sizeof(unsigned char);

    // Allocate device memory
    CUDA_CHECK(hipMalloc((void**)&d_src, srcSize));
    CUDA_CHECK(hipMalloc((void**)&d_dst, dstSize));

    // Copy data to device
    CUDA_CHECK(hipMemcpy(d_src, src, srcSize, hipMemcpyHostToDevice));

    float scaleX = (float)srcWidth / dstWidth;
    float scaleY = (float)srcHeight / dstHeight;

    // Define CUDA block and grid sizes
    dim3 blockSize(32, 32);  // 16x16 threads per block
    dim3 gridSize((dstWidth + blockSize.x - 1) / blockSize.x, (dstHeight + blockSize.y - 1) / blockSize.y);

    // Launch the CUDA kernel
    cuda_ResizeBicubicKernel << <gridSize, blockSize >> > (d_src, srcWidth, srcHeight, channels, d_dst, dstWidth, dstHeight, scaleX, scaleY);
    hipDeviceSynchronize();  // Ensure the kernel finishes before moving on


    // Copy the result back to host
    CUDA_CHECK(hipMemcpy(dst, d_dst, dstSize, hipMemcpyDeviceToHost));

    // Free device memory
    CUDA_CHECK(hipFree(d_src));
    CUDA_CHECK(hipFree(d_dst));
}
